#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <string>

#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#include "image-pnm.h"


#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

#define CHANNELS 3 // we have 3 channels corresponding to RGB
#define B 1



  // The input image is encoded as unsigned characters [0, 255]
  
  __global__ void colorConvert(unsigned char * grayImage,
                              unsigned char * rgbImage,
                              int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
      // get 1D coordinate for the grayscale image
      int grayOffset = y*width + x;
      // one can think of the RGB image having
      // CHANNEL times columns than the gray scale image
      int rgbOffset = grayOffset*CHANNELS;
      unsigned char r = rgbImage[rgbOffset ]; // red value for pixel
      unsigned char g = rgbImage[rgbOffset + 1]; // green value for pixel
      unsigned char b = rgbImage[rgbOffset+ 2]; // blue value for pixel
      // perform the rescaling and store it
      // We multiply by floating point constants
      grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }

  }

    __global__ void colorConvert(unsigned char * grayImage,
                              unsigned char * rgbImage,
                              int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
      // get 1D coordinate for the grayscale image
      int minX = x - B < 0 : 0 ? x - B,
          minY = y - B < 0 : 0 ? x - B,
          maxX = x + B > width : width ? x + B,
          maxY = y + B > height : height ? y + B;
        


      // perform the rescaling and store it
      // We multiply by floating point constants
      grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }

  }

int main(int argc, char** argv){
    size_t width = 512, height = 512;
    Image<float> in = Image<float>("P2", width, height);
    in.read("./image-grey.pgm");

    Image<float> out = Image<float>("P2", width, height);

    float* outData = out.getData();
    float* inData = in.getData();
    for (unsigned int i = 0; i < width ; i ++ ){
       for (unsigned int j = 0; j < height ; j ++){
          outData[i * width + j] = inData[i * width + j];
          
      }
    }
    out.write("copy.pgm");



  return 0;
}
