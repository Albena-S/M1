
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <stdlib.h>
#include <assert.h>

#define cudaCheckError() {                                                                       \
        hipError_t e=hipGetLastError();                                                        \
        if(e!=hipSuccess) {                                                                     \
            printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

__global__ void vecAddKernel(float *A,
                             float *B,
                             float *C,
                             int n){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if (i < n) C[i]=A[i] + B[i];
}

__global__ void vecAdd2DKernel(float *A,
                             float *B,
                             float *C,
                             int x,
			                       int y){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y; 	
  if (i < x && j < y) C[i*x + j]= A[i*x + j] + B[i*x + j];//C[j*y +i]= A[j*y +i] + B[j*y +i];
}

__host__ void vecAdd(float *h_A,
                       float *h_B,
                       float *h_C,
                       int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice );
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice );

  
  int threads=128;
  int blocks=(n+threads-1)/threads;


  vecAddKernel<<<blocks,threads>>>(d_A, d_B, d_C, n);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost );
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}

__host__ void vecAdd2D(float *h_A,
                       float *h_B,
                       float *h_C,
                       int x,
		                   int y) {
  int size = x*y * sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice );
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice );


  dim3 DimGrid((x-1)/16 + 1, (y-1)/16 + 1, 1);
  dim3 DimBlock(16, 16, 1);
  //le max c'est 1024 donc 32x32 mais c'est mieux d'utiliser 16x16
  //1. decision du nombre de threads donc dimblock
  //2.divise par le nombre de threads pour avoir le nombre de blocs donc dimGrid
   // nvcc deviceQuery.cpp -I/usr/local/cuda-11.4/samples/common/inc
   // ./a.out 


  //Maximum number of threads per multiprocessor:  2048
  //Maximum number of threads per block:           1024



  vecAdd2DKernel<<<DimGrid,DimBlock>>>(d_A, d_B, d_C, x, y);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost );
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}


void vecAddClassique(float *A,
                     float *B,
                     float *C,
                     int n){
  for (int i = 0; i < n; ++i) C[i]=A[i] + B[i];
}



int main(int argc, char** argv){
/*assert(argc==2);
  int n = atoi(argv[1]);
*/
/*  
  //2.5

  int n = 1024;
  float *h_A = new float[n];
  float *h_B = new float[n];
  float *h_C = new float[n]; 

  for (int i = 0; i < n; ++i){
    h_A[i] = i;
  }

  vecAdd(h_A, h_B, h_C, n); 

  for (int i = 0; i < n ; i++)
    std::cout << h_C[i] << " ";

  cudaCheckError();*/

  /*
  //2.6
  float *h_A = new float[n];
  float *h_B = new float[n];
  float *h_C = new float[n]; 

  for (int i = 0; i < n; ++i){
    h_A[i] = rand() % 65535;;
  }

  vecAddClassique(h_A, h_B, h_C, n); 

  for (int i = 0; i < n ; i++)
    std::cout << h_C[i] << " ";
  std::cout << std::endl;
  */
  //2.9
  int x = 32,y=32 ; 
  float *h_A = new float[x*y];
  float *h_B = new float[x*y];
  float *h_C = new float[x*y]; 


  for (int i = 0; i < x*y; ++i){
    h_A[i] = rand() % 65535;
    //std::cout << h_A[i] << " ";
  }
  std::cout << std::endl;
  

  vecAdd2D(h_A, h_B, h_C, x, y); 

  for (int i = 0; i < x*y ; i++)
    std::cout << h_C[i] << " ";
  std::cout << std::endl;


  return 0;
}
